#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include "CUDAPCISPHSolver3Example.hpp"

#include <Core/CUDA/CUDAArray.hpp>
#include <Core/CUDA/CUDAPCISPHSolver3.hpp>
#include <Core/PointGenerator/GridPointGenerator3.hpp>
#include <Core/Utils/Serialization.hpp>

#include <hip/hip_runtime.h>
#include <pystring/pystring.h>

#include <fstream>

using namespace CubbyFlow;

void SaveParticleAsPos(const CUDASPHSystemData3* particles,
                       const std::string& rootDir, int frameCnt)
{
    Array1<Vector3D> positions(particles->NumberOfParticles());
    ConstCUDAArrayView1<float4> cudaPositions = particles->Positions();
    for (std::size_t i = 0; i < particles->NumberOfParticles(); ++i)
    {
        const float4 position = cudaPositions.At(i);
        positions(i) = Vector3D{ static_cast<double>(position.x),
                                 static_cast<double>(position.y),
                                 static_cast<double>(position.z) };
    }
    char baseName[256];
    snprintf(baseName, sizeof(baseName), "frame_%06d.pos", frameCnt);
    std::string fileName = pystring::os::path::join(rootDir, baseName);
    std::ofstream file(fileName.c_str(), std::ios::binary);
    if (file)
    {
        printf("Writing %s...\n", fileName.c_str());
        std::vector<uint8_t> buffer;
        Serialize<Vector3D>(positions.View(), &buffer);
        file.write(reinterpret_cast<char*>(buffer.data()), buffer.size());
        file.close();
    }
}

void SaveParticleAsXYZ(const CUDASPHSystemData3* particles,
                       const std::string& rootDir, int frameCnt)
{
    Array1<Vector3D> positions(particles->NumberOfParticles());
    ConstCUDAArrayView1<float4> cudaPositions = particles->Positions();
    for (std::size_t i = 0; i < particles->NumberOfParticles(); ++i)
    {
        const float4 position = cudaPositions.At(i);
        positions(i) = Vector3D{ static_cast<double>(position.x),
                                 static_cast<double>(position.y),
                                 static_cast<double>(position.z) };
    }
    char baseName[256];
    snprintf(baseName, sizeof(baseName), "frame_%06d.xyz", frameCnt);
    std::string filename = pystring::os::path::join(rootDir, baseName);
    std::ofstream file(filename.c_str());
    if (file)
    {
        printf("Writing %s...\n", filename.c_str());
        for (const auto& pt : positions)
        {
            file << pt.x << ' ' << pt.y << ' ' << pt.z << std::endl;
        }
        file.close();
    }
}

void RunSimulation(const std::string& rootDir,
                   const CUDAPCISPHSolver3Ptr& solver, int numberOfFrames,
                   const std::string& format, double fps)
{
    const auto particles = solver->SPHSystemData();

    for (Frame frame(0, 1.0 / fps); frame.index < numberOfFrames; ++frame)
    {
        solver->Update(frame);

        if (format == "xyz")
        {
            SaveParticleAsXYZ(particles, rootDir, frame.index);
        }
        else if (format == "pos")
        {
            SaveParticleAsPos(particles, rootDir, frame.index);
        }
    }
}

void CUDAPCISPHSolver3Example::RunExample(const std::string& rootDir,
                                          float targetSpacing,
                                          int numberOfFrames,
                                          const std::string& format, double fps)
{
    // Setup solver
    auto solver = CUDAPCISPHSolver3::Builder().MakeShared();
    solver->SetDragCoefficient(0.0f);
    solver->SetRestitutionCoefficient(0.0f);
    solver->SetViscosityCoefficient(0.1f);
    solver->SetPseudoViscosityCoefficient(10.0f);
    solver->SetIsUsingFixedSubTimeSteps(true);
    solver->SetNumberOfFixedSubTimeSteps(1);

    auto particles = solver->SPHSystemData();
    particles->SetTargetSpacing(targetSpacing);
    particles->SetRelativeKernelRadius(1.8f);

    // Seed particles
    BoundingBox3D vol(Vector3D(), Vector3D(0.5, 0.5, 0.5));
    vol.Expand(-targetSpacing);
    Array1<Vector3D> rawPoints;
    GridPointGenerator3 generator;
    generator.Generate(vol, targetSpacing, &rawPoints);
    Array1<float4> hostData(rawPoints.Length());
    for (std::size_t i = 0; i < rawPoints.Length(); ++i)
    {
        Vector3F rp = rawPoints[i].CastTo<float>();
        hostData[i] = make_float4(rp[0], rp[1], rp[2], 0.0f);
    }
    CUDAArray1<float4> deviceData(hostData);
    particles->AddParticles(deviceData);
    printf("Number of particles: %zu\n", particles->NumberOfParticles());

    // Run simulation
    RunSimulation(rootDir, solver, numberOfFrames, format, fps);
}
