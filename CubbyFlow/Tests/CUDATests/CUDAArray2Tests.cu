// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include "doctest_proxy.hpp"

#include <Core/Array/ArrayView.hpp>
#include <Core/CUDA/CUDAArray.hpp>
#include <Core/CUDA/CUDAArrayView.hpp>

using namespace CubbyFlow;

TEST_CASE("[CUDAArray2] - Constructors")
{
    {
        CUDAArray2<float> arr;
        CHECK_EQ(0u, arr.Width());
        CHECK_EQ(0u, arr.Height());
    }
    {
        CUDAArray2<float> arr(CUDAStdArray<size_t, 2>(3, 7));
        CHECK_EQ(3u, arr.Width());
        CHECK_EQ(7u, arr.Height());
        for (size_t i = 0; i < 21; ++i)
        {
            CHECK_EQ(0.f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr(CUDAStdArray<size_t, 2>(1, 9), 1.5f);
        CHECK_EQ(1u, arr.Width());
        CHECK_EQ(9u, arr.Height());
        for (size_t i = 0; i < 9; ++i)
        {
            CHECK_EQ(1.5f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr(5, 2);
        CHECK_EQ(5u, arr.Width());
        CHECK_EQ(2u, arr.Height());
        for (size_t i = 0; i < 10; ++i)
        {
            CHECK_EQ(0.f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr(3, 4, 7.f);
        CHECK_EQ(3u, arr.Width());
        CHECK_EQ(4u, arr.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ(7.f, arr[i]);
        }
    }
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CHECK_EQ(4u, arr.Width());
        CHECK_EQ(3u, arr.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr[i]);
        }
    }
    {
        Array2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(arr);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(arr);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArrayView2<float> arrVew(arr.data(), arr.Size());
        CHECK_EQ(4u, arrVew.Width());
        CHECK_EQ(3u, arrVew.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arrVew[i]);
        }
    }
}

TEST_CASE("[CUDAArray2] - At")
{
    {
        float values[12] = { 0.f, 1.f, 2.f, 3.f, 4.f,  5.f,
                             6.f, 7.f, 8.f, 9.f, 10.f, 11.f };
        CUDAArray2<float> arr(4, 3);
        for (size_t i = 0; i < 12; ++i)
        {
            arr[i] = values[i];
        }

        // Test row-major
        CHECK_EQ(0.f, arr(0, 0));
        CHECK_EQ(1.f, arr(1, 0));
        CHECK_EQ(2.f, arr(2, 0));
        CHECK_EQ(3.f, arr(3, 0));
        CHECK_EQ(4.f, arr(0, 1));
        CHECK_EQ(5.f, arr(1, 1));
        CHECK_EQ(6.f, arr(2, 1));
        CHECK_EQ(7.f, arr(3, 1));
        CHECK_EQ(8.f, arr(0, 2));
        CHECK_EQ(9.f, arr(1, 2));
        CHECK_EQ(10.f, arr(2, 2));
        CHECK_EQ(11.f, arr(3, 2));
    }
}

TEST_CASE("[CUDAArray2] - CopyFrom")
{
    // From Array
    {
        Array2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From ArrayView
    {
        Array2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr.View());
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArray
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArrayView
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr2.CopyFrom(arr.View());
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }
}

TEST_CASE("[CUDAArray2] - CopyTo")
{
    // To Array
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        Array2<float> arr2(2, 5);

        arr.CopyTo(arr2);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // To ArrayView
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        Array2<float> arr2(4, 3);
        ArrayView2<float> arrView2 = arr2.View();

        arr.CopyTo(arrView2);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArray
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(2, 5);

        arr.CopyTo(arr2);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }

    // From CUDAArrayView
    {
        CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                                { 5.f, 6.f, 7.f, 8.f },
                                { 9.f, 10.f, 11.f, 12.f } });
        CUDAArray2<float> arr2(4, 3);
        CUDAArrayView2<float> arrView2 = arr2.View();

        arr.CopyTo(arrView2);
        CHECK_EQ(4u, arr2.Width());
        CHECK_EQ(3u, arr2.Height());
        for (size_t i = 0; i < 12; ++i)
        {
            CHECK_EQ((float)i + 1.f, arr2[i]);
        }
    }
}

TEST_CASE("[CUDAArray2] - Fill")
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });

    arr.Fill(42.0f);
    CHECK_EQ(4u, arr.Width());
    CHECK_EQ(3u, arr.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ(42.0f, arr[i]);
    }
}

TEST_CASE("[CUDAArray2] - Resize")
{
    {
        CUDAArray2<float> arr;
        arr.Resize(CUDAStdArray<size_t, 2>(2, 9));
        CHECK_EQ(2u, arr.Width());
        CHECK_EQ(9u, arr.Height());
        for (size_t i = 0; i < 18; ++i)
        {
            CHECK_EQ(0.f, arr[i]);
        }

        arr.Resize(CUDAStdArray<size_t, 2>(8, 13), 4.f);
        hipDeviceSynchronize();
        CHECK_EQ(8u, arr.Width());
        CHECK_EQ(13u, arr.Height());
        for (size_t i = 0; i < 8; ++i)
        {
            for (size_t j = 0; j < 13; ++j)
            {
                if (i < 2 && j < 9)
                {
                    CHECK_EQ(0.f, arr(i, j));
                }
                else
                {
                    CHECK_EQ(4.f, arr(i, j));
                }
            }
        }
    }
    {
        CUDAArray2<float> arr;
        arr.Resize(7, 6);
        CHECK_EQ(7u, arr.Width());
        CHECK_EQ(6u, arr.Height());
        for (size_t i = 0; i < 42; ++i)
        {
            CHECK_EQ(0.f, arr[i]);
        }

        arr.Resize(1, 9, 3.f);
        CHECK_EQ(1u, arr.Width());
        CHECK_EQ(9u, arr.Height());
        for (size_t i = 0; i < 1; ++i)
        {
            for (size_t j = 0; j < 9; ++j)
            {
                if (j < 6)
                {
                    CHECK_EQ(0.f, arr(i, j));
                }
                else
                {
                    CHECK_EQ(3.f, arr(i, j));
                }
            }
        }
    }
}

TEST_CASE("[CUDAArray2] - Clear")
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });

    arr.Clear();
    CHECK_EQ(0u, arr.Width());
    CHECK_EQ(0u, arr.Height());
}

TEST_CASE("[CUDAArray2] - Swap")
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
    CUDAArray2<float> arr2(2, 5, 42.f);

    arr.Swap(arr2);

    CHECK_EQ(2u, arr.Width());
    CHECK_EQ(5u, arr.Height());
    for (size_t i = 0; i < 10; ++i)
    {
        CHECK_EQ(42.0f, arr[i]);
    }

    CHECK_EQ(4u, arr2.Width());
    CHECK_EQ(3u, arr2.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ((float)i + 1.f, arr2[i]);
    }
}

TEST_CASE("[CUDAArray2] - View")
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });

    auto view = arr.View();

    CHECK_EQ(4u, view.Width());
    CHECK_EQ(3u, view.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ((float)i + 1.f, view[i]);
    }
    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            CHECK_EQ(arr(i, j), view(i, j));
        }
    }

    const auto& arrRef = arr;
    auto constView = arrRef.View();

    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ((float)i + 1.f, constView[i]);
    }
    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            CHECK_EQ(arr(i, j), constView(i, j));
        }
    }

    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            view(i, j) = float(i + 4 * j);
        }
    }

    for (size_t j = 0; j < 3; ++j)
    {
        for (size_t i = 0; i < 4; ++i)
        {
            CHECK_EQ(float(i + 4 * j), arr(i, j));
            CHECK_EQ(float(i + 4 * j), constView(i, j));
        }
    }
}

TEST_CASE("[CUDAArray2] - AssignmentOperator")
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
    CUDAArray2<float> arr2(2, 5, 42.f);

    arr2 = arr;

    CHECK_EQ(4u, arr.Width());
    CHECK_EQ(3u, arr.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ((float)i + 1.f, arr[i]);
    }

    CHECK_EQ(4u, arr2.Width());
    CHECK_EQ(3u, arr2.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ((float)i + 1.f, arr2[i]);
    }
}

TEST_CASE("[CUDAArray2] - MoveOperator")
{
    CUDAArray2<float> arr({ { 1.f, 2.f, 3.f, 4.f },
                            { 5.f, 6.f, 7.f, 8.f },
                            { 9.f, 10.f, 11.f, 12.f } });
    CUDAArray2<float> arr2(2, 5, 42.f);

    arr2 = std::move(arr);

    CHECK_EQ(0u, arr.Width());
    CHECK_EQ(0u, arr.Height());
    CHECK_EQ(nullptr, arr.data());
    CHECK_EQ(4u, arr2.Width());
    CHECK_EQ(3u, arr2.Height());
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ((float)i + 1.f, arr2[i]);
    }
}