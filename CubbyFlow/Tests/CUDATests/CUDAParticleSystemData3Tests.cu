#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include "doctest_proxy.hpp"

#include <Core/CUDA/CUDAParticleSystemData3.hpp>
#include <Core/CUDA/CUDAUtils.hpp>

using namespace CubbyFlow;

namespace
{
Vector4F MakeVector4F(float4 f)
{
    return Vector4F{ f.x, f.y, f.z, f.w };
}

struct ForEachCallback
{
    ForEachCallback(int* cnt) : count(cnt)
    {
        // Do nothing
    }

    template <typename Index, typename Float4>
    CUBBYFLOW_CUDA_HOST_DEVICE void operator()(size_t i, Float4 o, Index j,
                                               Float4 pt)
    {
        (void)o, (void)j, (void)pt;
        count[i] += 1;
    }

    int* count;
};

}  // namespace

TEST_CASE("[CUDAParticleSystemData3] - Constructors")
{
    CUDAParticleSystemData3 particleSystem;
    CHECK_EQ(0u, particleSystem.NumberOfParticles());

    CUDAParticleSystemData3 particleSystem2(100);
    CHECK_EQ(100u, particleSystem2.NumberOfParticles());

    size_t a0 = particleSystem2.AddFloatData(2.0f);
    size_t a1 = particleSystem2.AddFloatData(9.0f);
    size_t a2 = particleSystem2.AddVectorData({ 1.0f, -3.0f, 5.0f, 4.0f });
    size_t a3 = particleSystem2.AddIntData(8);

    CUDAParticleSystemData3 particleSystem3(particleSystem2);
    CHECK_EQ(100u, particleSystem3.NumberOfParticles());
    auto as0 = particleSystem3.FloatDataAt(a0);
    for (size_t i = 0; i < 100; ++i)
    {
        CHECK_EQ(2.0f, as0[i]);
    }

    auto as1 = particleSystem3.FloatDataAt(a1);
    for (size_t i = 0; i < 100; ++i)
    {
        CHECK_EQ(9.0f, as1[i]);
    }

    auto as2 = particleSystem3.VectorDataAt(a2);
    for (size_t i = 0; i < 100; ++i)
    {
        float4 f = as2[i];
        CHECK_EQ(1.0f, f.x);
        CHECK_EQ(-3.0f, f.y);
        CHECK_EQ(5.0f, f.z);
        CHECK_EQ(4.0f, f.w);
    }

    auto as3 = particleSystem3.IntDataAt(a3);
    for (size_t i = 0; i < 100; ++i)
    {
        CHECK_EQ(8, as3[i]);
    }
}

TEST_CASE("[CUDAParticleSystemData3] - Resize")
{
    CUDAParticleSystemData3 particleSystem;
    particleSystem.Resize(12);

    CHECK_EQ(12u, particleSystem.NumberOfParticles());
}

TEST_CASE("[CUDAParticleSystemData3] - AddFloatData")
{
    CUDAParticleSystemData3 particleSystem;
    particleSystem.Resize(12);

    size_t a0 = particleSystem.AddFloatData(2.0f);
    size_t a1 = particleSystem.AddFloatData(9.0f);

    CHECK_EQ(12u, particleSystem.NumberOfParticles());
    CHECK_EQ(0u, a0);
    CHECK_EQ(1u, a1);

    auto as0 = particleSystem.FloatDataAt(a0);
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ(2.0f, as0[i]);
    }

    auto as1 = particleSystem.FloatDataAt(a1);
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ(9.0f, as1[i]);
    }
}

TEST_CASE("[CUDAParticleSystemData3] - AddVectorData")
{
    CUDAParticleSystemData3 particleSystem;
    particleSystem.Resize(12);

    size_t a0 = particleSystem.AddVectorData(Vector4F(2.0f, 4.0f, -1.0f, 9.0f));
    size_t a1 = particleSystem.AddVectorData(Vector4F(9.0f, -2.0f, 5.0f, 7.0f));

    CHECK_EQ(12u, particleSystem.NumberOfParticles());
    CHECK_EQ(2u, a0);
    CHECK_EQ(3u, a1);

    auto as0 = particleSystem.VectorDataAt(a0);
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ(Vector4F(2.0f, 4.0f, -1.0f, 9.0f), MakeVector4F(as0[i]));
    }

    auto as1 = particleSystem.VectorDataAt(a1);
    for (size_t i = 0; i < 12; ++i)
    {
        CHECK_EQ(Vector4F(9.0f, -2.0f, 5.0f, 7.0f), MakeVector4F(as1[i]));
    }
}

TEST_CASE("[CUDAParticleSystemData3] - AddParticles")
{
    CUDAParticleSystemData3 particleSystem;
    particleSystem.Resize(12);

    particleSystem.AddParticles(
        Array1<Vector4F>({ Vector4F(1.0f, 2.0f, 3.0f, 4.0f),
                           Vector4F(4.0f, 5.0f, 6.0f, 7.0f) })
            .View(),
        Array1<Vector4F>({ Vector4F(7.0f, 8.0f, 9.0f, 10.0f),
                           Vector4F(8.0f, 7.0f, 6.0f, 5.0f) })
            .View());

    CHECK_EQ(14u, particleSystem.NumberOfParticles());
    auto p = particleSystem.Positions();
    auto v = particleSystem.Velocities();

    CHECK_EQ(Vector4F(1.0f, 2.0f, 3.0f, 4.0f), MakeVector4F(p[12]));
    CHECK_EQ(Vector4F(4.0f, 5.0f, 6.0f, 7.0f), MakeVector4F(p[13]));
    CHECK_EQ(Vector4F(7.0f, 8.0f, 9.0f, 10.0f), MakeVector4F(v[12]));
    CHECK_EQ(Vector4F(8.0f, 7.0f, 6.0f, 5.0f), MakeVector4F(v[13]));
}

TEST_CASE("[CUDAParticleSystemData3] - BuildNeighborSearcher")
{
    CUDAParticleSystemData3 particleSystem;
    Array1<Vector4F> positions = {
        Vector4F{ 0.1f, 0.0f, 0.4f, 0.0f }, Vector4F{ 0.6f, 0.2f, 0.6f, 0.0f },
        Vector4F{ 1.0f, 0.3f, 0.4f, 0.0f }, Vector4F{ 0.9f, 0.2f, 0.2f, 0.0f },
        Vector4F{ 0.8f, 0.4f, 0.9f, 0.0f }, Vector4F{ 0.1f, 0.6f, 0.2f, 0.0f },
        Vector4F{ 0.8f, 0.0f, 0.5f, 0.0f }, Vector4F{ 0.9f, 0.8f, 0.2f, 0.0f },
        Vector4F{ 0.3f, 0.5f, 0.2f, 0.0f }, Vector4F{ 0.1f, 0.6f, 0.6f, 0.0f },
        Vector4F{ 0.1f, 0.2f, 0.1f, 0.0f }, Vector4F{ 0.2f, 0.0f, 0.0f, 0.0f },
        Vector4F{ 0.2f, 0.6f, 0.1f, 0.0f }, Vector4F{ 0.1f, 0.3f, 0.7f, 0.0f },
        Vector4F{ 0.9f, 0.7f, 0.6f, 0.0f }, Vector4F{ 0.4f, 0.5f, 0.1f, 0.0f },
        Vector4F{ 0.1f, 0.1f, 0.6f, 0.0f }, Vector4F{ 0.7f, 0.8f, 1.0f, 0.0f },
        Vector4F{ 0.6f, 0.9f, 0.4f, 0.0f }, Vector4F{ 0.7f, 0.7f, 0.0f, 0.0f }
    };
    particleSystem.AddParticles(positions.View());

    float radius = 0.4f;
    particleSystem.BuildNeighborSearcher(radius);

    auto searcher = particleSystem.NeighborSearcher();
    Vector4F o{ 0.1f, 0.2f, 0.3f, 0.0f };
    CUDAArray1<float4> searchOrigin(1, ToFloat4(o));
    CUDAArray1<int> count(1, 0);
    searcher->ForEachNearbyPoint(searchOrigin.View(), radius,
                                 ForEachCallback(count.data()));

    int ans = 0;
    for (auto p : positions)
    {
        if ((p - o).Length() <= radius)
        {
            ans++;
        }
    }

    CHECK_EQ(ans, count[0]);
}

TEST_CASE("[CUDAParticleSystemData3] - BuildNeighborLists")
{
    CUDAParticleSystemData3 particleSystem;
    Array1<Vector4F> positions = {
        Vector4F{ 0.1f, 0.0f, 0.4f, 0.0f }, Vector4F{ 0.6f, 0.2f, 0.6f, 0.0f },
        Vector4F{ 1.0f, 0.3f, 0.4f, 0.0f }, Vector4F{ 0.9f, 0.2f, 0.2f, 0.0f },
        Vector4F{ 0.8f, 0.4f, 0.9f, 0.0f }, Vector4F{ 0.1f, 0.6f, 0.2f, 0.0f },
        Vector4F{ 0.8f, 0.0f, 0.5f, 0.0f }, Vector4F{ 0.9f, 0.8f, 0.2f, 0.0f },
        Vector4F{ 0.3f, 0.5f, 0.2f, 0.0f }, Vector4F{ 0.1f, 0.6f, 0.6f, 0.0f },
        Vector4F{ 0.1f, 0.2f, 0.1f, 0.0f }, Vector4F{ 0.2f, 0.0f, 0.0f, 0.0f },
        Vector4F{ 0.2f, 0.6f, 0.1f, 0.0f }, Vector4F{ 0.1f, 0.3f, 0.7f, 0.0f },
        Vector4F{ 0.9f, 0.7f, 0.6f, 0.0f }, Vector4F{ 0.4f, 0.5f, 0.1f, 0.0f },
        Vector4F{ 0.1f, 0.1f, 0.6f, 0.0f }, Vector4F{ 0.7f, 0.8f, 1.0f, 0.0f },
        Vector4F{ 0.6f, 0.9f, 0.4f, 0.0f }, Vector4F{ 0.7f, 0.7f, 0.0f, 0.0f }
    };
    particleSystem.AddParticles(positions.View());

    float radius = 0.4f;
    particleSystem.BuildNeighborSearcher(radius);
    particleSystem.BuildNeighborLists(radius);

    Array1<size_t> ansNeighborStarts(positions.Length());
    Array1<size_t> ansNeighborEnds(positions.Length());

    for (size_t i = 0; i < positions.Length(); ++i)
    {
        size_t cnt = 0;

        for (size_t j = 0; j < positions.Length(); ++j)
        {
            if (i != j && (positions[i] - positions[j]).Length() <= radius)
            {
                ++cnt;
            }
        }

        ansNeighborStarts[i] = cnt;
    }

    ansNeighborEnds[0] = ansNeighborStarts[0];
    for (size_t i = 1; i < ansNeighborStarts.Length(); ++i)
    {
        ansNeighborEnds[i] = ansNeighborEnds[i - 1] + ansNeighborStarts[i];
    }
    for (size_t i = 0; i < ansNeighborStarts.Length(); ++i)
    {
        ansNeighborStarts[i] = ansNeighborEnds[i] - ansNeighborStarts[i];
    }

    auto cuNeighborStarts = particleSystem.NeighborStarts();
    auto cuNeighborEnds = particleSystem.NeighborEnds();

    for (size_t i = 0; i < ansNeighborStarts.Length(); ++i)
    {
        CHECK_EQ(ansNeighborStarts[i], cuNeighborStarts[i]);
        CHECK_EQ(ansNeighborEnds[i], cuNeighborEnds[i]);
    }

    auto cuNeighborLists = particleSystem.NeighborLists();
    for (size_t i = 0; i < ansNeighborStarts.Length(); ++i)
    {
        size_t start = ansNeighborStarts[i];
        size_t end = ansNeighborEnds[i];
        for (size_t jj = start; jj < end; ++jj)
        {
            size_t j = cuNeighborLists[jj];
            CHECK_LE((positions[i] - positions[j]).Length(), radius);
        }
    }
}