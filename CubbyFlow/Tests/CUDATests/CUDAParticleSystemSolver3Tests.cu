#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include "doctest_proxy.hpp"

#include <Core/CUDA/CUDAParticleSystemSolver3.hpp>

using namespace CubbyFlow;

TEST_CASE("[CUDAParticleSystemSolver3 - PerfectBounce]")
{
    CUDAParticleSystemSolver3 solver;
    solver.SetDragCoefficient(0.0);
    solver.SetRestitutionCoefficient(1.0);

    auto particles = solver.ParticleSystemData();
    particles->AddParticle({ 0.0f, 3.0f, 0.0f, 0.0f },
                           { 1.0f, 0.0f, 0.0f, 0.0f });

    Array1<float> x(1000);
    Array1<float> y(1000);

    Frame frame;
    frame.timeIntervalInSeconds = 1.0 / 300.0;
    for (; frame.index < 1000; frame.Advance())
    {
        solver.Update(frame);

        float4 pos = particles->Positions()[0];
        x[frame.index] = pos.x;
        y[frame.index] = pos.y;
    }
}