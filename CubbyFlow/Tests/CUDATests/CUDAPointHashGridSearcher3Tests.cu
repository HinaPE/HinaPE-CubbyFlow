#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include "doctest_proxy.hpp"

#include <Core/Array/Array.hpp>
#include <Core/CUDA/CUDAArray.hpp>
#include <Core/CUDA/CUDAPointHashGridSearcher3.hpp>
#include <Core/Searcher/PointParallelHashGridSearcher.hpp>

using namespace CubbyFlow;

namespace
{
struct ForEachCallback
{
    CUBBYFLOW_CUDA_HOST_DEVICE void operator()(size_t i, float4 o, size_t j,
                                               float4 pt)
    {
        (void)i, (void)o;
        visited[j] = 1;

        if (j == 1)
        {
            isValid[j] = 0;
            return;
        }

        if (j == 0)
        {
            isValid[j] = points[0] == pt;
        }
        else if (j == 2)
        {
            isValid[j] = points[2] == pt;
        }
    }

    float4* points;
    int* isValid;
    int* visited;
};
}  // namespace

TEST_CASE("[CUDAPointHashGridSearcher3] - Build")
{
    // CPU baseline
    Array1<Vector3D> points = { Vector3D(0, 1, 3), Vector3D(2, 5, 4),
                                Vector3D(-1, 3, 0) };

    PointParallelHashGridSearcher3 searcher({ 4, 4, 4 }, std::sqrt(10.0f));
    searcher.Build(points);

    // GPU
    CUDAArray1<float4> pointsD(3);
    pointsD[0] = make_float4(0, 1, 3, 0);
    pointsD[1] = make_float4(2, 5, 4, 0);
    pointsD[2] = make_float4(-1, 3, 0, 0);

    CUDAPointHashGridSearcher3 searcherD({ 4, 4, 4 }, std::sqrt(10.0f));
    searcherD.Build(pointsD.View());

    // Compare
    CHECK_EQ(searcher.Keys().Length(), searcherD.Keys().Length());
    CHECK_EQ(searcher.StartIndexTable().Length(),
             searcherD.StartIndexTable().Length());
    CHECK_EQ(searcher.EndIndexTable().Length(),
             searcherD.EndIndexTable().Length());
    CHECK_EQ(searcher.SortedIndices().Length(),
             searcherD.SortedIndices().Length());

    for (size_t i = 0; i < searcher.Keys().Length(); ++i)
    {
        uint32_t valD = searcherD.Keys()[i];
        CHECK_EQ(searcher.Keys()[i], valD);
    }

    for (size_t i = 0; i < searcher.StartIndexTable().Length(); ++i)
    {
        uint32_t valD = searcherD.StartIndexTable()[i];
        if (valD == 0xffffffff)
        {
            CHECK_EQ(std::numeric_limits<size_t>::max(),
                     searcher.StartIndexTable()[i]);
        }
        else
        {
            CHECK_EQ(searcher.StartIndexTable()[i], valD);
        }
    }

    for (size_t i = 0; i < searcher.EndIndexTable().Length(); ++i)
    {
        uint32_t valD = searcherD.EndIndexTable()[i];
        if (valD == 0xffffffff)
        {
            CHECK_EQ(std::numeric_limits<size_t>::max(),
                     searcher.EndIndexTable()[i]);
        }
        else
        {
            CHECK_EQ(searcher.EndIndexTable()[i], valD);
        }
    }

    for (size_t i = 0; i < searcher.SortedIndices().Length(); ++i)
    {
        size_t valD = searcherD.SortedIndices()[i];
        CHECK_EQ(searcher.SortedIndices()[i], valD);
    }
}

TEST_CASE("[CUDAPointHashGridSearcher3] - ForEachNearbyPoint")
{
    CUDAArray1<float4> pointsD(3);
    pointsD[0] = make_float4(0, 1, 3, 0);
    pointsD[1] = make_float4(2, 5, 4, 0);
    pointsD[2] = make_float4(-1, 2.9f, 0, 0);

    CUDAArray1<float4> origins(1, make_float4(0, 0, 0, 0));
    CUDAArray1<int> isValid(3, 1);
    CUDAArray1<int> visited(3, 0);

    CUDAPointHashGridSearcher3 searcherD(4, 4, 4, std::sqrt(10.0f));
    searcherD.Build(pointsD.View());

    ForEachCallback func;
    func.points = pointsD.data();
    func.isValid = isValid.data();
    func.visited = visited.data();

    searcherD.ForEachNearbyPoint(origins.View(), std::sqrt(10.0f), func);

    int iv = isValid[0];
    int vd = visited[0];
    CHECK_EQ(1, iv);
    CHECK_EQ(1, vd);
    iv = isValid[1];
    vd = visited[1];
    CHECK_EQ(1, iv);
    CHECK_EQ(0, vd);
    iv = isValid[2];
    vd = visited[2];
    CHECK_EQ(1, iv);
    CHECK_EQ(1, vd);
}